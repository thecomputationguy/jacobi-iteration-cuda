#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <assert.h>
#include <iostream>
#include <chrono>
#include "solvers.cuh"
#include "solvers.cu"


int main(int arc, char* argv[])
{
    std::cout<<"\n** Starting Jacobi Solver **\n";
    const int resolution_gpu[5] = {10, 100, 1000, 2000, 3000};
    const int iterations = 1000;
    const int num_resolutions = 5;
    bool useGPU;
    int numBlocks = 1;
    int blockSize = 256;

    for(int i = 0; i < num_resolutions; i++)
    {
        const int resolution = resolution_gpu[i];

        // GPU code runs in this block
        useGPU = true;
        jacobiSolverGPU<float> jacobiGPU(resolution, useGPU, numBlocks, blockSize);
        std::cout<<"\nResolution : "<<resolution<<std::endl;
        auto start = std::chrono::high_resolution_clock::now();
        for(int j = 0; j < iterations; j++)
        {
            auto result = jacobiGPU.solve();
        }
        auto stop = std::chrono::high_resolution_clock::now();
        auto elapsed_gpu = std::chrono::duration_cast<std::chrono::microseconds>(stop - start) / iterations;

        // CPU code runs in this block
        useGPU = false;
        jacobiSolverCPU<float> jacobiCPU(resolution, useGPU);
        start = std::chrono::high_resolution_clock::now();
        for(int j = 0; j < iterations; j++)
        {
            auto result = jacobiCPU.solve();
        }
        stop = std::chrono::high_resolution_clock::now();
        auto elapsed_cpu = std::chrono::duration_cast<std::chrono::microseconds>(stop - start) / iterations;
        
        std::cout<<"\tCPU : "<<elapsed_cpu.count()<<" microseconds"<<std::endl;
        std::cout<<"\tGPU : "<<elapsed_gpu.count()<<" microseconds"<<std::endl;
    }

    return 0;
}