#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include <hip/hip_runtime.h>
#include <assert.h>
#include <iostream>
// #include "include/jacobi_cpu.h"
// #include "include/jacobi_gpu.cuh"

// void jacobiCPU(float* x_new, const float* A, float* x_current, float* b, const int Nx, const int Ny, const int iterations)
// {
//     int i, j;
//     float sum;

//     for(i = 0; i < Nx; i++)
//     {
//         sum = 0.0;
//         for(j = 0; j < Ny; j++)
//         {
//             if(i != j)
//                 sum += A[i * Ny + j] * x_current[j];
//         }
//         x_new[i] = (b[i] - sum) / A[i * Ny + i];
//     }
// }

__global__
void jacobiGPUBasic(float* x_new, float* A, float* x_current, float* b, const int Nx, const int Ny)
{
    float sum = 0.0;
    int idx = threadIdx.x;
    int j;

    for(j = 0; j < Ny; j++)
    {
        if(idx != j)
        {
            sum += A[idx * Ny + j] * x_current[j];
        }
        x_new[idx] = (b[idx] - sum) / A[idx * Ny + idx];
    }
}

void allocateHostMemory(float **x_current, float **x_next, float **b, float **A, int resolution)
{
    printf("\nAllocating Host Memory...");
    *x_current = (float*)malloc(resolution * sizeof(float));
    *x_next = (float*)malloc(resolution * sizeof(float));
    *b = (float*)malloc(resolution * sizeof(float));
    *A = (float*)malloc(resolution * resolution * sizeof(float));
    printf("\nHost Memory Allocated.\n");
}

void allocateDeviceMemory(float **x_current_device, float **x_next_device, float **b_device, float **A_device, 
                        int resolution,
                        float **x_current, float **x_next, float **b, float **A)
{
    printf("\nAllocating Device Memory...");
    assert(hipSuccess == hipMalloc((void**) x_current_device, resolution * sizeof(float)));
    assert(hipSuccess == hipMalloc((void**) x_next_device, resolution * sizeof(float)));
    assert(hipSuccess == hipMalloc((void**) b_device, resolution * sizeof(float)));
    assert(hipSuccess == hipMalloc((void**) A_device, resolution * resolution * sizeof(float) * sizeof(float)));
    printf("\nDevice Memory Allocated.\n");

    printf("\nCopying to Device Memory...");
    assert(hipSuccess == hipMemcpy(*x_current_device, *x_current, resolution * sizeof(float), hipMemcpyHostToDevice));
    assert(hipSuccess == hipMemcpy(*x_next_device, *x_next, resolution * sizeof(float), hipMemcpyHostToDevice));
    assert(hipSuccess == hipMemcpy(*b_device, *b, resolution * sizeof(float), hipMemcpyHostToDevice));
    assert(hipSuccess == hipMemcpy(*A_device, *A, resolution * resolution * sizeof(float), hipMemcpyHostToDevice));
    printf("\nCopied to Device.");
}

void freeDeviceMemory(float **x_current_device, float **x_next_device, float **b_device, float **A_device)
{
    assert(hipSuccess == hipFree(*A_device));
    assert(hipSuccess == hipFree(*x_current_device));
    assert(hipSuccess == hipFree(*x_next_device));
    assert(hipSuccess == hipFree(*b_device));
}

void freeHostMemory(float **x_current, float **x_next, float **b, float **A)
{
    free(*A);
    free(*x_current);
    free(*x_next);
    free(*b);
}

int main(int arc, char* argv[])
{
    unsigned int resolution = 10000;
    unsigned int increment = 10000;
    unsigned int final_resolution = 40000;
    unsigned int iterations = 50;
    clock_t start_time;
    clock_t end_time;
    double elapsed_time;
    int blockSize = ceil(resolution / 768);
    int numBlocks = 1;

    printf("\n** Starting Jacobi Solver on CPU **\n");
    // while(resolution <= final_resolution)
    // {
    //     float* x_current = (float*)malloc(resolution * sizeof(float));
    //     float* x_next = (float*)malloc(resolution * sizeof(float));
    //     float* b = (float*)malloc(resolution * sizeof(float));
    //     float* A = (float*)malloc(resolution * resolution * sizeof(float));

    //     start_time = clock();        
    //     jacobiCPU(x_next, A, x_current, b, resolution, resolution);
    //     end_time = clock();
    //     elapsed_time = (end_time - start_time) / CLOCKS_PER_SEC;

    //     printf("\nResolution       : %d", resolution);
    //     printf("\nIterations       : %d", iterations);
    //     printf("\nTime Elapsed (s) : %.2lf", elapsed_time / iterations);
    //     printf("\n");

    //     free(x_current);
    //     free(x_next);
    //     free(b);
    //     free(A);

    //     resolution += increment;
    // }

    printf("\n** Starting Jacobi Solver on GPU (Basic) **\n");
    const int resolution_gpu[5] = {10, 100, 1000, 10000, 15000};
    iterations = 1000;
    std::cout<<"\n trying C++"<<std::endl;

    for (int iter = 0; iter < 5; iter++)
    {
        const int resolution = resolution_gpu[iter];
        
        float *x_current_device, *x_next_device, *b_device, *A_device;
        float *x_current, *x_next, *b, *A;
        allocateHostMemory(&x_current, &x_next, &b, &A, resolution);

        allocateDeviceMemory(&x_current_device, &x_next_device, &b_device, &A_device,
                            resolution,
                            &x_current, &x_next, &b, &A);

        start_time = clock();
        for(int i = 0; i < iterations; i++)
        {
            jacobiGPUBasic<<<numBlocks, blockSize>>>(x_next_device, A_device, x_current_device, b_device, resolution, resolution);
        }
        end_time = clock();
        elapsed_time = (end_time - start_time) ;

        printf("\nResolution       : %d", resolution);
        printf("\nIterations       : %d", iterations);
        printf("\nTime Elapsed (s) : %lf", elapsed_time);
        printf("\n");

        freeDeviceMemory(&x_current_device, &x_next_device, &b_device, &A_device);
        freeHostMemory(&x_current, &x_next, &b, &A);        
    }

    return 0;
}