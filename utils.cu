#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <assert.h>
#include <hip/hip_runtime_api.h>
#include "utils.cuh"

template <typename T>
hostCUDAVariable<T>::hostCUDAVariable(const size_t size, const bool useGPU) : size_(size), useGPU_(useGPU)
{
    x_ = (T*)malloc(size_ * sizeof(T));

    if(useGPU_)
    {
        assert(hipSuccess == hipMalloc((void**) &xd_, size_ * sizeof(T)));
    }
    
}
       
template <typename T>
void hostCUDAVariable<T>::copyToDevice()
{
    assert(hipSuccess == hipMemcpy(xd_, x_, size_ * sizeof(T), hipMemcpyHostToDevice));
}

template <typename T>
void hostCUDAVariable<T>::copyToHost()
{
    assert(hipSuccess == hipMemcpy(x_, xd_, size_ * sizeof(T), hipMemcpyDeviceToHost));
}

template <typename T>
T*&  hostCUDAVariable<T>::getDeviceVariable()
{
    return xd_;
}

template <typename T>
T*&  hostCUDAVariable<T>::getHostVariable()
{
    return x_;
}

template <typename T>
hostCUDAVariable<T>::~hostCUDAVariable()
{
    if(useGPU_) 
    {
        hipFree(xd_);
    }              

    free(x_);
}

template<typename T>
Solver<T>::Solver(const size_t size, const bool useGPU) : A_(size * size, useGPU), b_(size, useGPU), 
                                                        x_current_(size, useGPU), x_next_(size, useGPU), 
                                                        resolution_(size)
{
}

template<typename T>
T*& Solver<T>::solve()
{
}
